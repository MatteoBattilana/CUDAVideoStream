#include "hip/hip_runtime.h"

#include <stdio.h>
#include <fcntl.h>    /* For O_RDWR */
#include <sys/ioctl.h>
#include <unistd.h>   /* For open(), creat() */
#include <netdb.h>
#include <sys/epoll.h>
#include <sys/socket.h>
#include <unistd.h>
#include <signal.h>
#include <sys/stat.h>
#include <sys/types.h>
#include <hip/hip_runtime.h>
#include <>
#include <time.h>
#include <stdint.h>
#include "opencv2/opencv.hpp"
#include <pthread.h>
#include "v4l.h"
#include <cmath>

#define K 3
#define C 3
#define TILE_SIZE 10
#define BLOCK_SIZE (TILE_SIZE + K - 1)
#define NSTREAMS 1
#define GPU
//#define FILTER
#define HEATMAP

using namespace cv;

struct mat_ready {
    Mat *pframe;
    int *h_xs;
    unsigned int h_pos;
};

struct ctxs {
    VideoCapture *cap;
    int cap_w_fd;
    Mat *sampleMat;
    int show_r_fd;
    int ptr_w_fd;
    int ptr_r_fd;
    int proc_w_fd;
    int proc_w_fd_B;
    // unsigned int **phpos;
    // int **pxs;
};

#ifdef GPU
__constant__ float dev_k[K*K];

__global__ void convolution_kernel(uint8_t *current, uint8_t *filtered, int width, int height)
{
    __shared__ uint8_t N_ds[BLOCK_SIZE][BLOCK_SIZE*C];

    int tx = threadIdx.x;   
    int ty = threadIdx.y;   
    int row_o = blockIdx.y*TILE_SIZE + ty;
    int col_o = blockIdx.x*TILE_SIZE + tx;
    int row_i = row_o - K/2;
    int col_i = col_o - K/2;

    if(row_i >= 0 && row_i < height && col_i >= 0 && col_i < width){
        N_ds[ty][tx*C] = current[row_i*width*C + col_i*C];
        N_ds[ty][tx*C+1] = current[row_i*width*C + col_i*C + 1];
        N_ds[ty][tx*C+2] = current[row_i*width*C + col_i*C + 2];
    } else {
        N_ds[ty][tx*C] = 0;
        N_ds[ty][tx*C+1] = 0;
        N_ds[ty][tx*C+2] = 0;
    }

    __syncthreads();


    for (int color = 0; color < C; color++){
        int output = 0;
        if(ty < TILE_SIZE && tx < TILE_SIZE){
            for(int i = 0; i < K; i++)
                for(int j = 0; j < K; j++){
                    output += dev_k[i*K+j] * N_ds[i+ty][(j+tx)*C+color];
                }

            if(row_o < height && col_o < width){
                filtered[row_o*width*C + col_o*C + color] = output;
            }
        }
    }
}

__device__ float maxF(float a, float b){
    return a > b? a : b;
}

__global__ void kernel(uint8_t *current, uint8_t *previous, uint8_t *diff, int maxSect, unsigned int *pos, int *xs, uint8_t *d_heat_pixels) {
    int x = threadIdx.x + blockDim.x * blockIdx.x;
    unsigned int npos;
    int df;

    int max = x * maxSect + maxSect;
    for (int i = x * maxSect; i < max; i++) {

        df = current[i] - previous[i];
        if (df < -20 || df > 20) {
            npos = atomicInc(pos, 6220801);
            diff[npos] = df;
            xs[npos] = i;
        } else {
            current[i] -= df;
        }

    #ifdef HEATMAP
        if(i%3==0){
            float ddf = fabsf(current[i] - previous[i]) + fabsf(current[i+1] - previous[i+1]) + fabsf(current[i+2] - previous[i+2]);
            float diff1 = ddf/(255.0*2.0);
            float r = fminf(fmaxf(sinf(M_PI*diff1 - M_PI/2.0)*255.0, 0.0),255.0);
            float g = fminf(fmaxf(sinf(M_PI*diff1)*255.0, 0.0),255.0);
            float b = fminf(fmaxf(sinf(M_PI*diff1 + M_PI/2.0)*255.0, 0.0),255.0);
            d_heat_pixels[i] = b;
            d_heat_pixels[i+1] = g;
            d_heat_pixels[i+2] = r;
        }
    #endif

    }
}
#endif

void *th_cap_hdl(void *args) {

    struct mat_ready *pready;
    struct ctxs *pctx = (struct ctxs *)args;

    while(1) {
        read(pctx->ptr_r_fd, &pready, sizeof pready);
        *pctx->cap >> *(pready->pframe);

        write(pctx->cap_w_fd, &pready, sizeof pready);
    }

    return NULL;
}

void *th_show_hdl(void *args) {
    struct epoll_event ev, events[10];
    struct addrinfo *result, *rp;
    int sfd, epollfd, nfds, sfd2;
    struct mat_ready *pready;
    bool skip = true;

    getaddrinfo("127.0.0.1", "2734", NULL, &result);
    for (rp = result; rp != NULL; rp = rp->ai_next) {
        if ((sfd = socket(AF_INET, SOCK_STREAM, 0)) == -1) {
            continue;
        }

        if (bind(sfd, rp->ai_addr, rp->ai_addrlen) != -1) {
            break;
        }

        close(sfd);
        perror("MOH!");
    }

    epollfd = epoll_create1(0);
    ev.events = EPOLLIN;
    ev.data.fd = sfd;
    epoll_ctl(epollfd, 1, sfd, &ev);

    if (listen(sfd, 10) < 0) {
        perror("OH!");
        exit(errno);
    }

    nfds = epoll_wait(epollfd, events, 10, -1);

    for (int i = 0; i < nfds; i++) {
        if (events[i].data.fd == sfd) {
            sfd2 = accept(sfd, NULL, NULL);
            if (sfd2 < 0) perror("ACCEPT");
        }
    }

    struct ctxs *pctx = (struct ctxs *)args;
    int tot = 3 * pctx->sampleMat->cols * pctx->sampleMat->rows;

    // uint8_t *mem = new uint8_t[sizeof **pctx->phpos + tot * sizeof **pctx->phpos + tot];
    write(sfd2, pctx->sampleMat->data, tot);
    // Mat previous = pctx->sampleMat->clone();

    while(1) {
        read(pctx->show_r_fd, &pready, sizeof pready);
        // printf("show on %p\n", pframe);

        if (skip ^= 1) {
            skip = true;
        }

        // memcpy(mem, *pctx->phpos, sizeof **pctx->phpos);
        // memcpy(mem + sizeof **pctx->phpos, pready->h_xs, **pctx->phpos * sizeof *pready->h_xs);
        // memcpy(mem + sizeof **pctx->phpos + **pctx->phpos * sizeof *pready->h_xs, pready->pframe->data, **pctx->phpos);
        // printf("Writing all, xs %ld\n", **pctx->phpos * sizeof **pctx->pxs);
        // for (int i = 0; i < 10; i++) {
        //     printf(" ## xs %i = %d\n", (*(pctx->pxs))[i], pframe->data[i]);
        // }

        // for (int i = 0; i < 10; i++) {
        //     printf(" -- xs %i = %d\n", (mem + sizeof **pctx->phpos)[i], (mem + sizeof **pctx->phpos + **pctx->phpos * sizeof **pctx->pxs)[i]);
        // }

        int ret = write(sfd2, &pready->h_pos, sizeof pready->h_pos);
        if (ret != sizeof pready->h_pos) {
            perror("write1");
        }

        ret = write(sfd2, pready->h_xs, pready->h_pos * sizeof *pready->h_xs);
        if (ret != pready->h_pos * sizeof *pready->h_xs) {
            perror("write2");
        }

        ret = write(sfd2, pready->pframe->data, pready->h_pos);
        if (ret != pready->h_pos) {
            perror("write3");
        }


        write(pctx->ptr_w_fd, &pready, sizeof pready);
    }

    return NULL;
}

float* computeGaussianKernel(float sigma){
    double sum = 0;
    float* k = (float*)malloc(K*K*sizeof(float));
    for (int i = 0; i < K; i++){
        for (int j = 0; j < K; j++){
            double x = i - (K - 1) / 2.0;
            double y = j - (K - 1) / 2.0;
            k[i*K+j] = (1.0/(2.0*M_PI*sigma*sigma)) * exp(-((x*x + y*y)/(2.0*sigma*sigma)));
            sum += k[i*K+j];
        }
    }

    for (int i = 0; i < K; i++) {
        for (int j = 0; j < K; j++) {
            k[i*K+j] /= sum;
        }
    }

    return k;
}

int main() {
    // Initialize kernel for filter
    float * k = computeGaussianKernel(2);

    VideoCapture cap;
    if (!cap.open(0, CAP_V4L2)) return 1;
    auto codec = cv::VideoWriter::fourcc('M','J','P','G');
    cap.set(cv::CAP_PROP_FOURCC, codec);


    cap.set(3, 1920);
    cap.set(4, 1080);

    Mat base;
    cap >> base;

    int cap_pipe[2];
    int show_pipe[2];
    int ptr_pipe[2];
    int fork_pipe[2];
    int fork_pipe_B[2];
    pipe(cap_pipe);
    pipe(show_pipe);
    pipe(ptr_pipe);
    pipe(fork_pipe);
    pipe(fork_pipe_B);

    pthread_mutex_t fifosize_mtx;
    pthread_mutex_init(&fifosize_mtx, NULL);

    int *h_xs;
    unsigned int *h_pos;
    struct ctxs ctx = { 
        .cap = &cap,
        .cap_w_fd = cap_pipe[1], 
        .sampleMat = &base ,
        .show_r_fd = show_pipe[0],
        .ptr_w_fd = ptr_pipe[1],
        .ptr_r_fd = ptr_pipe[0],
        .proc_w_fd = fork_pipe[1],
        .proc_w_fd_B = fork_pipe_B[1]
    };

    pthread_t th_cap;
    pthread_t th_show;
    pthread_create(&th_cap, NULL, th_cap_hdl, &ctx);
    pthread_create(&th_show, NULL, th_show_hdl, &ctx);

    struct mat_ready *pready;
    for (int i = 0; i < 20; i++) {

        pready = new struct mat_ready;

#ifdef GPU
        // Copy filter kernel to memory
        hipMemcpyToSymbol(HIP_SYMBOL(dev_k), k, K*K * sizeof(float) );
        
        uint8_t *h_frame;
        hipHostMalloc((void **)&h_frame, 3 * ctx.sampleMat->rows * ctx.sampleMat->cols * sizeof *h_frame);
        hipHostMalloc((void **)&pready->h_xs, 3 * ctx.sampleMat->rows * ctx.sampleMat->cols * sizeof *pready->h_xs);
        pready->pframe = new Mat(ctx.sampleMat->rows, ctx.sampleMat->cols, ctx.sampleMat->type(), h_frame);
#else
        pready->pframe = new Mat(ctx.sampleMat->rows, ctx.sampleMat->cols, ctx.sampleMat->type());
        pready->h_xs = new int[3 * ctx.sampleMat->rows * ctx.sampleMat->cols];
#endif

        pready->h_pos = 0;
        write(ctx.ptr_w_fd, &pready, sizeof pready);
    }

    int total = 3 * ctx.sampleMat->rows * ctx.sampleMat->cols;

#ifdef CPU
    uint8_t *h_diff = new uint8_t[total];
    h_xs = new int[total];
    h_pos = new unsigned int[1];
#elif defined(GPU)
    struct hipDeviceProp_t prop;
    uint8_t *d_current, *d_previous;
    uint8_t *d_diff;
    uint8_t *d_filtered;
    uint8_t *d_heat_pixels;
    int *d_xs;
    unsigned int *d_pos;
    hipStream_t streams[4];

    hipGetDeviceProperties(&prop, 0);
    hipStreamCreate(&streams[0]);
    hipStreamCreate(&streams[1]);
    hipStreamCreate(&streams[2]);
    hipStreamCreate(&streams[3]);

    hipMalloc((void **)&d_diff, total * sizeof *d_diff);
    hipMalloc((void **)&d_xs, total * sizeof *d_xs);
    hipMalloc((void **)&d_current, total * sizeof *d_current);
    hipMalloc((void **)&d_previous, total * sizeof *d_previous);
    hipMalloc((void **)&d_filtered, total * sizeof *d_filtered);
    hipMalloc((void **)&d_heat_pixels, total * sizeof *d_heat_pixels);

    hipMalloc((void **)&d_pos, sizeof *d_pos);
    // hipMemset((void *)d_pos, 0, sizeof *d_pos);

    uint8_t *h_diff, *h_diff2, *h_current;
    int *h_xs2;
    hipHostMalloc((void **)&h_diff, total * sizeof *h_diff);
    hipHostMalloc((void **)&h_diff2, total * sizeof *h_diff2);
    hipHostMalloc((void **)&h_current, total * sizeof *h_current);
    hipHostMalloc((void **)&h_pos, sizeof *h_pos);
    hipHostMalloc((void **)&h_xs, total * sizeof *h_xs);
    hipHostMalloc((void **)&h_xs2, total * sizeof *h_xs2);

    int maxAtTime = total / prop.maxThreadsPerBlock;
    hipMemcpy(d_current, ctx.sampleMat->data, total * sizeof *ctx.sampleMat->data, hipMemcpyHostToDevice);
    hipMemcpy(d_filtered, ctx.sampleMat->data, total * sizeof *ctx.sampleMat->data, hipMemcpyHostToDevice);

    int tot4 = total / 1;
    int max4 = maxAtTime / 1;
    uint8_t *dcurr4_0 = d_current;
    uint8_t *dcurr4_1 = d_current + tot4;
    uint8_t *dcurr4_2 = d_current + tot4;
    uint8_t *dcurr4_3 = d_current + 3*tot4;
    uint8_t *dprev4_0 = d_previous;
    uint8_t *dprev4_1 = d_previous + tot4;
    uint8_t *dprev4_2 = d_previous + tot4;
    uint8_t *dprev4_3 = d_previous + 3*tot4;
    uint8_t *ddiff_0 = d_diff;
    uint8_t *dfiltered_0 = d_filtered;
    uint8_t *ddiff_1 = d_diff + tot4;
    uint8_t *ddiff_2 = d_diff + tot4;
    uint8_t *ddiff_3 = d_diff + 3*tot4;
    // uint8_t *pframe_0 = pframe->data;
    // uint8_t *pframe_1 = pframe->data + total/4;
    // uint8_t *pframe_2 = pframe->data + total/2;
    // uint8_t *pframe_3 = pframe->data + 3*total/4;

    // Setup filter grid
    dim3 blockSize, gridSize;
    blockSize.x = BLOCK_SIZE, blockSize.y = BLOCK_SIZE, blockSize.z = 1;
    gridSize.x = ceil((float)ctx.sampleMat->cols/TILE_SIZE),
    gridSize.y = ceil((float)ctx.sampleMat->rows/TILE_SIZE),
    gridSize.z = 1;
#endif

    Mat previous = ctx.sampleMat->clone();
    Mat heatMap = ctx.sampleMat->clone();

    auto begin0 = std::chrono::high_resolution_clock::now();
    while (1) {
        auto begin = std::chrono::high_resolution_clock::now();

        auto begin2 = std::chrono::high_resolution_clock::now();
        read(cap_pipe[0], &pready, sizeof pready);
        auto end2 = std::chrono::high_resolution_clock::now();

        auto begin3 = std::chrono::high_resolution_clock::now();
#ifdef CPU
        Mat pvs = pready->pframe->clone();

        *h_pos = 0;
        for (int i = total-1; i >= 0; i--) {
            pready->pframe->data[*h_pos] = pready->pframe->data[i] - previous.data[i];
            pready->h_xs[*h_pos] = i;
            (*h_pos)++;
        }

        previous = pvs;

#elif defined(GPU)

    #ifdef FILTER
        uint8_t *d_prev = d_filtered;
        d_filtered = dprev4_0;
        dprev4_0 = d_prev;
    #else
        uint8_t *d_prev = dcurr4_0;
        dcurr4_0 = dprev4_0;
        dprev4_0 = d_prev;
    #endif

        d_prev = dcurr4_1;
        dcurr4_1 = dprev4_1;
        dprev4_1 = d_prev;

        // d_prev = d_current;
        // d_current = d_previous;
        // d_previous = d_prev;

        // d_prev = dcurr4_1;
        // dcurr4_1 = dprev4_1;
        // dprev4_1 = d_prev;

        // d_prev = dcurr4_2;
        // dcurr4_2 = dprev4_2;
        // dprev4_2 = d_prev;

        // d_prev = dcurr4_3;
        // dcurr4_3 = dprev4_3;
        // dprev4_3 = d_prev;


        // Apply filter
        hipMemcpyAsync(dcurr4_0, pready->pframe->data, tot4, hipMemcpyHostToDevice, streams[0]);
    
    #ifdef FILTER
        convolution_kernel<<<gridSize, blockSize>>>(dcurr4_0, d_filtered, ctx.sampleMat->cols, ctx.sampleMat->rows);
        //kern_test<<<1, prop.maxThreadsPerBlock, 0, streams[0]>>>(dcurr4_0, d_filtered, max4);
        hipDeviceSynchronize();

        hipMemset(d_pos, 0, sizeof *d_pos);
        kernel<<<1, prop.maxThreadsPerBlock, 0, streams[0]>>>(d_filtered, dprev4_0, ddiff_0, max4, d_pos, d_xs, d_heat_pixels);
    #else
        hipMemset(d_pos, 0, sizeof *d_pos);
        kernel<<<1, prop.maxThreadsPerBlock, 0, streams[0]>>>(dcurr4_0, dprev4_0, ddiff_0, max4, d_pos, d_xs, d_heat_pixels);
    #endif

        hipMemcpyAsync(pready->pframe->data, ddiff_0, tot4, hipMemcpyDeviceToHost, streams[0]);//TODO: *h_pos instead of tot4
        #ifdef HEATMAP
        hipMemcpyAsync(heatMap.data, d_heat_pixels, tot4, hipMemcpyDeviceToHost, streams[0]);//TODO: *h_pos instead of tot4
        #endif
        hipMemcpyAsync(pready->h_xs, d_xs, tot4 * sizeof *d_xs, hipMemcpyDeviceToHost, streams[0]);

        // hipMemcpyAsync(dcurr4_1, pready->pframe->data + tot4, tot4, hipMemcpyHostToDevice, streams[1]);
        // kernel<<<1, prop.maxThreadsPerBlock, 0, streams[1]>>>(dcurr4_1, dprev4_1, ddiff_1, max4, d_pos, d_xs);
        // hipMemcpyAsync(pready->pframe->data + tot4, ddiff_1, tot4, hipMemcpyDeviceToHost, streams[1]);//TODO: *h_pos instead of tot4
        // hipMemcpyAsync(pready->h_xs + tot4, d_xs + tot4, tot4 * sizeof *d_xs, hipMemcpyDeviceToHost, streams[1]);


        // hipMemcpyAsync(dcurr4_1, pframe_1, tot4, hipMemcpyHostToDevice, streams[1]);
        // kernel<<<1, prop.maxThreadsPerBlock, 0, streams[1]>>>(dcurr4_1, dprev4_1, ddiff_1, max4, d_pos);
        // hipMemcpyAsync(pframe_1, ddiff_1, tot4, hipMemcpyDeviceToHost, streams[1]);

        // hipMemcpyAsync(dcurr4_2, pframe_2, tot4, hipMemcpyHostToDevice, streams[2]);
        // kernel<<<1, prop.maxThreadsPerBlock, 0, streams[2]>>>(dcurr4_2, dprev4_2, ddiff_2, max4, d_pos);
        // hipMemcpyAsync(pframe_2, ddiff_2, tot4, hipMemcpyDeviceToHost, streams[2]);

        // hipMemcpyAsync(dcurr4_3, pframe_3, tot4, hipMemcpyHostToDevice, streams[3]);
        // kernel<<<1, prop.maxThreadsPerBlock, 0, streams[3]>>>(dcurr4_3, dprev4_3, ddiff_3, max4, d_pos);
        // hipMemcpyAsync(pframe_3, ddiff_3, tot4, hipMemcpyDeviceToHost, streams[3]);

        hipDeviceSynchronize();
        hipMemcpy(&pready->h_pos, d_pos, sizeof *d_pos, hipMemcpyDeviceToHost); 

        #ifdef HEATMAP
        namedWindow("ht", WINDOW_GUI_NORMAL);
        imshow("ht", heatMap);
        if (waitKey(10) == 27) break;  // stop capturing by pressing ESC
        #endif

#endif
        auto end3 = std::chrono::high_resolution_clock::now();

        write(show_pipe[1], &pready, sizeof pready);

        auto end = std::chrono::high_resolution_clock::now();

        auto end0 = std::chrono::high_resolution_clock::now();
        if (std::chrono::duration_cast<std::chrono::nanoseconds>(end0 - begin0).count() > 1e9) {
            begin0 = end0;

            auto elaps = std::chrono::duration_cast<std::chrono::nanoseconds>(end - begin);
            auto elaps2 = std::chrono::duration_cast<std::chrono::nanoseconds>(end2 - begin2);
            auto elaps3 = std::chrono::duration_cast<std::chrono::nanoseconds>(end3 - begin3);

            float unit = 1 / ((float)elaps.count() * 1e-9);
            printf("\rFPS: %5.0f\tFOR: %5.2f ms\tREAD: %9.2f\tPOS: %7d\t BW: %5d kbps", unit, (float)elaps3.count() * 1e-6, (float)elaps2.count() * 1e-6, pready->h_pos, (int)((pready->h_pos<<4)*unit*1e-3));
            fflush(stdout);
        }

    }

    return 0;
}