#include "hip/hip_runtime.h"
#include <stdio.h>
#include <fcntl.h>    /* For O_RDWR */
#include <sys/ioctl.h>
#include <unistd.h>   /* For open(), creat() */
#include <netdb.h>
#include <sys/epoll.h>
#include <sys/socket.h>
#include <unistd.h>
#include <signal.h>
#include <sys/stat.h>
#include <sys/types.h>
#include <hip/hip_runtime.h>
#include <>
#include <time.h>
#include <stdint.h>
#include "opencv2/opencv.hpp"
#include <pthread.h>
#include <cmath>
#include <cstdlib>
using namespace cv;

#define H 1080
#define W 1920
#define C 3

__global__ void kernel(uint8_t *current, uint8_t *previous, int maxSect, uint8_t* d_heat_pixels) {
    int x = threadIdx.x + blockDim.x * blockIdx.x;
    int start = x * maxSect;
    int max = start + maxSect;

    for (int i = start; i < max; i=i+3) {
        // Compute the pixel difference
		int pixelDiff = fabsf(current[i] - previous[i]) + fabsf(current[i+1] - previous[i+1]) + fabsf(current[i+2] - previous[i+2]); 
		float diff1 = pixelDiff/(255*3.0);
		
		// Map different into the three color component
		int r = fminf(fmaxf(sinf(M_PI*diff1 - M_PI/2.0)*255.0, 0.0),255.0);
		int g = fminf(fmaxf(sinf(M_PI*diff1)*255.0, 0.0),255.0);
		int b = fminf(fmaxf(sinf(M_PI*diff1 + M_PI/2.0)*255.0, 0.0),255.0);
		d_heat_pixels[i] = b;
		d_heat_pixels[i+1] = g;
		d_heat_pixels[i+2] = r;
    }
}
   

int main(int argc, char *argv[]) {
    int threads = 1024;
    if(argc == 2){
        threads = atoi(argv[1]);
    }
    printf("Number of threads set to: %d\n", threads);

    uint8_t *d_current, *d_previous;
    uint8_t *d_heat_pixels;

    hipMalloc((void **)&d_current, W*H*C * sizeof *d_current);
    hipMalloc((void **)&d_previous, W*H*C * sizeof *d_previous);
    hipMalloc((void **)&d_heat_pixels, W*H*C * sizeof *d_heat_pixels);

    Mat image1, image2, res;
    VideoCapture cap;
    if (!cap.open("/dev/video0")) return 1;
    auto codec = cv::VideoWriter::fourcc('M','J','P','G');
    cap.set(cv::CAP_PROP_FOURCC, codec);
    cap.set(3, W);
    cap.set(4, H);
    cap >> image1;
    res = image1.clone();
    hipMemcpy(d_previous, image1.data,  W*H*C * sizeof *image1.data, hipMemcpyHostToDevice);

    auto start = std::chrono::high_resolution_clock::now();
    auto end = std::chrono::high_resolution_clock::now();
    for (int a = 0; a < 100; a++){
        cap >> image2;
        
        namedWindow("Original", WINDOW_GUI_NORMAL);
        imshow("Original", image2);
        if (waitKey(10) == 27) {
            break;  // stop capturing by pressing ESC
        }


        start = std::chrono::high_resolution_clock::now();

        hipMemcpy(d_current, image1.data,  W*H*C * sizeof *image2.data, hipMemcpyHostToDevice);
        hipMemcpy(d_current, image2.data,  W*H*C * sizeof *image2.data, hipMemcpyHostToDevice);
        kernel<<<1, threads>>>(d_current, d_previous, ((W*H*C)/threads), d_heat_pixels);
        hipMemcpy(res.data, d_heat_pixels, W*H*C * sizeof *res.data, hipMemcpyDeviceToHost);

        end = std::chrono::high_resolution_clock::now();
        auto elaps = std::chrono::duration_cast<std::chrono::nanoseconds>(end - start);
        printf("\rHeatmap time generation: %.3f ms", (float)elaps.count() * 1e-6);
        fflush(stdout);
        namedWindow("HeatMap", WINDOW_GUI_NORMAL);
        imshow("HeatMap", res);
        if (waitKey(10) == 27) {
            break;  // stop capturing by pressing ESC
        }

        image1 = image2.clone();
    }
    


    return 0;
}