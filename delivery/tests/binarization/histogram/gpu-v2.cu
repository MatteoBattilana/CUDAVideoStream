#include "hip/hip_runtime.h"
#include <iostream>
#include <opencv2/opencv.hpp>

using namespace std;
using namespace cv;

#define NAIVE

#define H 1080
#define W 1920

#define SHMEM_SIZE 256

__global__ void generate_histogram(uint8_t *grayscale, int *histogram, int maxSect) {

    __shared__ int shared_histogram[SHMEM_SIZE];
    int x = threadIdx.x + blockDim.x * blockIdx.x;
    int start = x * maxSect;
    int max = start + maxSect;
    int sum = 0;

    // Initialize shared memory

    if(threadIdx.x < 256){
        shared_histogram[threadIdx.x] = 0;
    }

    __syncthreads();


    for (int i = start; i < max; i++) {
        atomicAdd(&shared_histogram[grayscale[i]], 1);
    }

    __syncthreads();
    
    if(threadIdx.x < 256){
        atomicAdd(&histogram[threadIdx.x], shared_histogram[threadIdx.x]);
    }

}


int main(int argc, char const *argv[]) {

    // VideoCapture cap(video_file);
    VideoCapture cap(0, CAP_V4L2);

    if (!cap.isOpened())
        cerr << "Error opening video stream\n";

    auto codec = cv::VideoWriter::fourcc('M', 'J', 'P', 'G');
    cap.set(cv::CAP_PROP_FOURCC, codec);
    cap.set(3, W);
    cap.set(4, H);

    Mat frame, bw, binarize;
    frame.create(H, W, CV_8UC3);
    bw.create(H, W, CV_8UC1);
    binarize.create(H, W, CV_8UC1);
    int sum = 0;

    // GPU

    unsigned int *d_pos;

    struct hipDeviceProp_t prop;
    hipGetDeviceProperties(&prop, 0);
    int total_grayscale = W * H;
    int nMaxThreads = prop.maxThreadsPerBlock;
    int maxAtTime = total_grayscale / nMaxThreads;
    int *d_histogram;
    uint8_t *d_grayscale;

    hipMalloc((void **)&d_grayscale, total_grayscale * sizeof *d_grayscale);
    hipMalloc((void **)&d_histogram, 256 * sizeof(int));

    //

    while (1) {
        cap >> frame;
        if (frame.empty())
            return 0;

        imshow("input", frame);

        if (waitKey(10) == 27) {
            break; // stop capturing by pressing ESC
        }

        // generate black & white image

        for (int row = 0; row < H; row++) {
            for (int col = 0; col < W; col++) {
                bw.at<uchar>(row, col) = 0.114 * frame.at<Vec3b>(row, col)[0] + 0.587 * frame.at<Vec3b>(row, col)[1] + 0.299 * frame.at<Vec3b>(row, col)[2];
            }
        }

        imshow("bw", bw);
        // waitKey(0);
        if (waitKey(10) == 27) {
            break; // stop capturing by pressing ESC
        }

        // find the histogram of the occurency of the values from 0 to 255
        // Naive implementation is a for loop from 0 to 255 and then a loop inside on the matrix
        int histogram[256] = {0};
        int h_histogram[256];

#ifdef NAIVE

        // CPU
        auto start = std::chrono::high_resolution_clock::now();
        for (int row = 0; row < H; row++) {
            for (int col = 0; col < W; col++) {
                int index = bw.at<uchar>(row, col);
                histogram[index]++;
            }
        }

        // GPU

        hipMemcpy(d_grayscale, bw.data, total_grayscale, hipMemcpyHostToDevice);

        generate_histogram<<<1, nMaxThreads>>>(d_grayscale, d_histogram, maxAtTime);

        hipMemcpy(h_histogram, d_histogram, 256 * sizeof(int), hipMemcpyDeviceToHost);
        hipDeviceSynchronize();

        //

        for (int i = 0; i < 256; i++) {
            printf("%d \n", h_histogram[i]);
        }
        int trash;
        // scanf("%d", &trash);
        int max = -1, sec_max = -1;
        int index_max = -1, index_sec_max = -1;
        for (int i = 0; i < 256; i++) {
            if (h_histogram[i] >= max) {
                index_sec_max = index_max;
                index_max = i;
                max = h_histogram[i];
                sec_max = max;
            } else if (h_histogram[i] > sec_max && h_histogram[i] < max) {
                sec_max = h_histogram[i];
                index_sec_max = i;
            }
        }
        int threshold = (index_max + index_sec_max) / 2;
        if (threshold < 20)
            threshold = 20;
        printf("%d %d", index_max, index_sec_max);
        // scanf("%d", &trash);

        for (int row = 0; row < H; row++) {
            for (int col = 0; col < W; col++) {
                if (bw.at<uchar>(row, col) > threshold) {
                    binarize.at<uchar>(row, col) = 255;
                } else {
                    binarize.at<uchar>(row, col) = 0;
                }
            }
        }

        imshow("binarize", binarize);

        auto end = std::chrono::high_resolution_clock::now();
        auto elaps = std::chrono::duration_cast<std::chrono::nanoseconds>(end - start);
        printf("\rHeatmap time generation: %.3f ms", (float)elaps.count() * 1e-6);
        fflush(stdout);
#endif
        // Optimized implementation is a for loop inside on the matrix, and then value of the pixel is used to insert the value in the histogram
    }

    cap.release();
    return 0;
}